#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "../inc/lorena_cpu.cuh"


#define Min(x,y) (fabs(x) < fabs(y) ? fabs(x) : fabs(y))

#define EPSILON 5
#define PI 3.14159265358979323846

/****************************************************************/

void lorena_cpu::modifica_A_B(int *adjlist, double a, double b,double *A,double *B,int size) {
	for (int i=0; i<size; i++) {
		A[i] += (double) a*adjlist[i];
		B[i] += (double) b*adjlist[i];
	}
}

/****************************************************************/

double lorena_cpu::taglio(int **adjmat,int *r,int size) {
	register int i,j;
	double s=0;

	for (i=0; i<size-1; i++){
		for (j=i+1; j<size; j++){
			s += adjmat[i][j]*(1-r[i]*r[j]);
        }
    }

	return s/2;
}

/****************************************************************/

int *lorena_cpu::taglio_massimo(int **adjmat,double *teta,int size) {
	register int i,j,k;
	register double t,T=0;
	int *R;
    double max_alfa;
	double alfa;

	R = (int *) calloc(size,sizeof(int));
    int *res=(int*)malloc(size*sizeof(int));

	for (i=0; i<size; i++) {
		alfa = (teta[i] > PI) ? teta[i]-PI : teta[i];
		for (j=0; j<size; j++)
			R[j] = ((teta[j] >= alfa) && (teta[j] < alfa+PI)) ? 1 : -1;
		t = taglio(adjmat,R,size);
		if ( t > T ){
            T = t;
            max_alfa=alfa;
        }
            
	}
    
    /*Store the resulted partition and return the pointer*/
    for(j=0;j<size;j++)
        res[j] = ((teta[j] >= max_alfa) && (teta[j] < max_alfa+PI)) ? 1 : -1;
	std::cout<<"Lorena ->"<<T<<"\n";
    free(R);
    return res;
}

void lorena_cpu::mappa_cerchio_unitario(int **adjmat,double *teta,double *A,double *B,int size){
	int OK = 1, nround = 0;
	double p,alfa;
    
	while ( OK ) {
		nround++;
		OK = 0;
		for (int k=0; k<size; k++) {
			alfa = teta[k];
			teta[k] = atan(B[k]/A[k]); 
			if (A[k] >= 0) 
				teta[k] += PI; 
			else if (B[k] > 0) 
				teta[k] += 2*PI;
			modifica_A_B(adjmat[k], cos(teta[k])-cos(alfa), sin(teta[k])-sin(alfa),A,B,size);
			if ( Min(alfa-teta[k],2*PI-alfa+teta[k]) > EPSILON )
				OK = 1;
		}
	}
}

int *lorena_cpu::mapAndCut(int **adjmat,double *teta,int size){
    double *t=(double*)malloc(size*sizeof(double));
    /*Hard copy value of teta to avoid external changes*/
    memcpy(t,teta,size*sizeof(double));

    double *A=(double*)malloc(size*sizeof(double));
    double *B=(double*)malloc(size*sizeof(double));
    
	for (int i=0; i<size; i++) {
		A[i] = B[i] = 0;
		for(int j=0; j<size; j++) {
			A[i] += adjmat[i][j]*cos(teta[j]);
			B[i] += adjmat[i][j]*sin(teta[j]);
		}
	}
    
    lorena_cpu::mappa_cerchio_unitario(adjmat,t,A,B,size);
    //std::cout<<"Start cutting the mapped points\n";
    int *res=lorena_cpu::taglio_massimo(adjmat,t,size);
    free(A);
    free(B);
    return res;
}

/****************************************************************/

/*void lorena_cpu::inizializza_strutture(int argc, char *argv[]){
	register int i,j;
	char s[11];

	if ( argc < 3 ) {
		printf("Manca un parametro!\n");
		exit(1);
	}
	if ((in = fopen(argv[1],"r")) == NULL) {
		printf("File non trovato!\n");
		exit(1);
	}
	srand(atoi(argv[2]));  // seme fissato x ripetere stesso exp
	
	// srand(time(NULL)&RAND_MAX);
	N = atoi(fgets(s,10,in));
	adjmat = (Row *) malloc(N*sizeof(Row));
	teta = (double *) calloc(N,sizeof(double));
	A = (double *) calloc(N,sizeof(double));
	B = (double *) calloc(N,sizeof(double));
	for (i=0; i<N; i++)
		teta[i] = (double) 2*PI*rand()/RAND_MAX;
	for (i=0; i<N; i++) {
		adjmat[i] = (int *) malloc(N*sizeof(int));
		A[i] = B[i] = 0;
		for(j=0; j<N; j++) {
			adjmat[i][j] = fgetc(in)-48;
			A[i] += adjmat[i][j]*cos(teta[j]);
			B[i] += adjmat[i][j]*sin(teta[j]);
		}
		fgetc(in);
	}
	fclose(in);
}*/

/*Init with externally created input*/
/*void lorena_cpu::inizializza_strutture(int **adjmat,int size){
    N=size;
    //adjmat is supposed to be read-only input, so pointer can be assigned
    adjmat=adjmat;
    teta=(double*)malloc(size*sizeof(double));
    A=(double*)malloc(size*sizeof(double));
    B=(double*)malloc(size*sizeof(double));
    
    //vector teta,A,B are modified during the algorithm, so they must be hard-copied

	for (int i=0; i<N; i++)
		teta[i] = (double) 2*PI*rand()/RAND_MAX;
	for (int i=0; i<N; i++) {
		A[i] = B[i] = 0;
		for(int j=0; j<N; j++) {
			A[i] += adjmat[i][j]*cos(teta[j]);
			B[i] += adjmat[i][j]*sin(teta[j]);
            std::cout<<"Init lorena\n";
		}
	}
    std::cout<<"Init lorena\n";
}*/
/**************************************************************/

/******************************    Main     ******************************/

/*int main(int argc, char *argv[]) {
	register int k;
	int OK = 1, nround = 0;
	double p;

	inizializza_strutture(argc,argv);
	while ( OK ) {
		nround++;
		OK = 0;
		for (k=0; k<N; k++) {
			alfa = teta[k];
			teta[k] = atan(B[k]/A[k]); 
			if (A[k] >= 0) 
				teta[k] += PI; 
			else if (B[k] > 0) 
				teta[k] += 2*PI;
			modifica_A_B(k, cos(teta[k])-cos(alfa), sin(teta[k])-sin(alfa));
			if ( Min(alfa-teta[k],2*PI-alfa+teta[k]) > EPSILON )
				OK = 1;
		}
	}
	taglio_massimo();
	stampa_teta();
	printf("Num rounds = %d\n",nround);
	return(0);
}*/




