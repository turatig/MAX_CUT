#include<iostream>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "../inc/Graph.cuh"

Graph::Graph(int s,int p){
    size=s;   
    hipMallocManaged(&adjmat,size*sizeof(int*));
    
    srand(time(NULL));
    for(int i=0;i<size;i++){

        hipMallocManaged(&adjmat[i],size*sizeof(int));

        for(int j=0;j<size;j++){

            //Graph is supposed to be undirected, so adjacency matrix is symmetric
            if(j<i){ adjmat[i][j]=adjmat[j][i]; }

            else{

                if(rand()%100<p){  adjmat[i][j]=1; }
                else{ adjmat[i][j]=0; }

            }
        }
    }
}

Graph::Graph(char *argv) {
    char s[11];
    FILE *in;

    if( (in = fopen(argv,"r")) == NULL ) {
      printf("File non trovato!\n");
      exit(1);
    }

    size = atoi( fgets(s,10,in));
    std::cout<<"CREATING GRAPH OF SIZE "<<size<<"\n";
    hipMallocManaged(&adjmat,size*sizeof(int *));
   
    for (int i=0; i<size; i++) {
      hipMallocManaged(&adjmat[i],size*sizeof(int));

      for(int j=0; j<size; j++)
        adjmat[i][j] = fgetc(in)-48;
        fgetc(in);
    }
    fclose(in);
}
Graph::~Graph(){
    for(int i=0;i<size;i++)
        hipFree(adjmat[i]);
    hipFree(adjmat);
}
int ** Graph::getAdjmat(){ return adjmat;}
int Graph::getSize(){ return size;}

void Graph::print(){
    for(int i=0;i<size;i++){
        for(int j=0;j<size;j++)
            std::cout<<adjmat[i][j]<<" ";
        std::cout<<"\n";
    }
}
