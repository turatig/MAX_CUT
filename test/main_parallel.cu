#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <fstream>
#include "../inc/rete_cpu.cuh"
#include "../inc/rete_gpu.cuh"
#include "../inc/lorena_cpu.cuh"
#include "../inc/lorena_gpu.cuh"
#include "../inc/lorena_batch.cuh"
#include "../inc/Graph.cuh"
#include "../inc/utils.cuh"


#define PI 3.14159265358979323846

/*
Driver program to test parallel implementation of the algorithms against sequential one.
Call main size seed sparsity.
    -size: the number of nodes in the graph
    -seed: to reproduce the experiment
    -sparsity: percentage of nodes connected (given as integer parameter e.g. 50 means approximately 50% of nodes connected)
*/
int main(int argc,char **argv){
    double start,elapsed;
    long cost;
    int size, sparsity,seed;
    hipDeviceProp_t dev_prop;

    hipGetDeviceProperties(&dev_prop,0);
    std::cout<<"Zero-copy enabled: "<<dev_prop.canMapHostMemory<<"\n";
    
    std::ofstream hop_par_out,lor_par_out;

    hop_par_out.open("hopfield_parallel.txt",std::ios_base::app);
    lor_par_out.open("lorena_parallel.txt",std::ios_base::app);
  
    if(argc<2){
        std::cout<<"The size of the graph must be specified\n";
        return -1;
    }
    size=atoi(argv[1]);
    if(argc>=3){
        seed=atoi(argv[2]);
        srand(seed);
    }
    else{
        std::cout<<"Random seeded\n";
        seed=time(NULL);
        srand(seed);
    }
    if(argc>=4)
        sparsity=atoi(argv[3]);
    else
        sparsity=rand()%100;

    std::cout<<"Size: "<<size<<" Seed: "<<seed<<" Sparsity: "<<sparsity<<"\n";

    hop_par_out<<"Size: "<<size<<"\n";
    lor_par_out<<"Size: "<<size<<"\n";

    Graph *g=new Graph(size,sparsity);
    
    start=cpuSecond();
    int *status_gpu=stabilizeHopfieldNet(g);
    elapsed=cpuSecond()-start;
    std::cout<<"Hopfield: parallel implementation ended in "<<elapsed<<" sec\n";
    cost=lorena_cpu::taglio(g->getAdjmat(),status_gpu,g->getSize());
    std::cout<<"Hopfield->"<<cost<<"\n";
    hop_par_out<<"Time: "<<elapsed<<" Cost: "<<cost<<"\n";
    
    hipHostFree(status_gpu);
    
    double *teta=(double*)malloc(g->getSize()*sizeof(double));
	for (int i=0; i<g->getSize(); i++)
		teta[i] = (double) 2*PI*rand()/RAND_MAX;
    

    start=cpuSecond();
    double *updated_teta=circleMap(g,teta);
    elapsed=cpuSecond()-start;
    std::cout<<"Lorena--mapping points: parallel implementation ended in "<<elapsed<<" sec\n";
    lor_par_out<<"(Map)"<<" Time: "<<elapsed<<"\n";

    start=cpuSecond();
    status_gpu=maximumCut(g,updated_teta);
    elapsed=cpuSecond()-start;
    std::cout<<"Lorena--find best partition: parallel implementation ended in "<<elapsed<<" sec\n";
    cost=lorena_cpu::taglio(g->getAdjmat(),status_gpu,g->getSize());
    lor_par_out<<"(Cut)"<<" Time: "<<elapsed<<" Cost: "<<cost<<"\n";



    start=cpuSecond();
    status_gpu=maximumCutBatch(g,updated_teta,256);
    elapsed=cpuSecond()-start;
    std::cout<<"Lorena--find best partition: parallel batch implementation ended in "<<elapsed<<" sec\n";
    lor_par_out<<"(Cut_Batch)"<<" Time: "<<elapsed<<"\n";

    
    
    hop_par_out.close();
    lor_par_out.close();

    free(status_gpu);
    free(teta);
    free(updated_teta);
    delete g;

}
